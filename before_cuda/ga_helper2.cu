#include "hip/hip_runtime.h"
#include "ga_helper.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void calculateSumDistancesKernel(int *dPopulation, double *dDistanceMatrix, double *dSumDistances, int populationSize, int cityCount) {
    // todo: share memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < populationSize) {
        int* path = dPopulation + idx * cityCount;        
        double totalDistance = dDistanceMatrix[path[cityCount - 1]*cityCount + path[0]];
        for (int i = 0; i < cityCount - 1; ++i) {
            totalDistance += dDistanceMatrix[path[i]*cityCount + path[i + 1]];
        }
        dSumDistances[idx] = totalDistance;
    }
}

void calculateSumDistances2(int *dPopulation, double *dDistanceMatrix, double *dSumDistances, int populationSize, int cityCount, int threadsPerBlock) {
    int num_blocks = (populationSize + threadsPerBlock - 1) / threadsPerBlock;
    calculateSumDistancesKernel<<<num_blocks, threadsPerBlock>>>(dPopulation, dDistanceMatrix, dSumDistances, populationSize, cityCount);
    hipDeviceSynchronize();
}

__host__  double* calculateSumDistances(const int *hPopulation, double *hDistanceMatrix, int populationSize, int cityCount, int threadsPerBlock) {
    double *hSumDistances;
    hSumDistances = new double[populationSize];
    double *dSumDistances;
    int *dPopulation;
    double *dDistanceMatrix;
    hDistanceMatrix = new double[cityCount * cityCount];
    hipMalloc((void **)&dSumDistances, populationSize * sizeof(double));
    hipMalloc((void **)&dPopulation, populationSize * cityCount * sizeof(int));
    hipMemcpy(dPopulation, hPopulation, populationSize * cityCount * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&dDistanceMatrix, cityCount * cityCount * sizeof(double));
    hipMemcpy(dDistanceMatrix, hDistanceMatrix, cityCount * cityCount * sizeof(double), hipMemcpyHostToDevice);
    calculateSumDistances2(dPopulation, dDistanceMatrix, dSumDistances, populationSize, cityCount, threadsPerBlock);
    hipMemcpy(hSumDistances, dSumDistances, populationSize  * sizeof(double), hipMemcpyDeviceToHost);
    return hSumDistances;
}