#include "hip/hip_runtime.h"
#include "ga_helper.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void calculateDistanceMatrixKernel(const int *cityX, const int *cityY, double *dDistanceMatrix, int cityCount) {
    // todo: share memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < cityCount * cityCount) {
        int i = idx / cityCount;
        int j = idx % cityCount;
        int dx = cityX[i] - cityX[j];
        int dy = cityY[i] - cityY[j];
        dDistanceMatrix[idx] = sqrt((double)(dx * dx + dy * dy));
    }
}

void calculateDistanceMatrix2(const int *dCityX, const int *dCityY, double *dDistanceMatrix, int cityCount, int threadsPerBlock) {
    int num_blocks = (cityCount * cityCount + threadsPerBlock - 1) / threadsPerBlock;
    calculateDistanceMatrixKernel<<<num_blocks, threadsPerBlock>>>(dCityX, dCityY, dDistanceMatrix, cityCount);
    hipDeviceSynchronize();
}

__host__  double* calculateDistanceMatrix(const int *hCityX, const int *hCityY, int cityCount, int threadsPerBlock) {
    int *dCityX, *dCityY;
    double *hDistanceMatrix, *dDistanceMatrix;
    hDistanceMatrix = new double[cityCount * cityCount];
    hipMalloc((void **)&dCityX, cityCount * sizeof(int));
    hipMalloc((void **)&dCityY, cityCount * sizeof(int));
    hipMalloc((void **)&dDistanceMatrix, cityCount * cityCount * sizeof(double));
    hipMemcpy(dCityX, hCityX, cityCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dCityY, hCityY, cityCount * sizeof(int), hipMemcpyHostToDevice);
    calculateDistanceMatrix2(dCityX, dCityY, dDistanceMatrix, cityCount, threadsPerBlock);
    hipMemcpy(hDistanceMatrix, dDistanceMatrix, cityCount * cityCount * sizeof(double), hipMemcpyDeviceToHost);
    return hDistanceMatrix;
}