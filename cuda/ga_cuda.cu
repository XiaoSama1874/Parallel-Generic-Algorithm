#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <random>
#include <limits>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <string>

#define CUDA_CHECK(call)                                                                          \
    do                                                                                            \
    {                                                                                             \
        hipError_t err = call;                                                                   \
        if (err != hipSuccess)                                                                   \
        {                                                                                         \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
            exit(1);                                                                              \
        }                                                                                         \
    } while (0)

using namespace std;

int CITY_COUNT = 100;                 // city scale
int POPULATION_SIZE = 1000;           // population scale
int GENERATIONS = 1000;               // iteration times
const float MUTATION_RATE = 0.1;     // mutation rate
const float ELITISM_THRESHOLD = 0.2; // elite
bool PRINT_EACH_ITERATION = true;     // debug
const int TOURNAMENT_SIZE = 5;        // For tournament selection
struct City
{
    int x, y;
};

// Kernel to initialize hiprand states
__global__ void initRNGStates(hiprandState_t *states, unsigned long seed, int POPULATION_SIZE)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

// GPU kernel: compute fitness
__global__ void computeFitnessKernel(const int *population, const float *distanceMatrix, float *fitness, int POPULATION_SIZE, int CITY_COUNT)
{
    extern __shared__ float sharedDistanceMatrix[];
    int sharedMatrixSize = CITY_COUNT * CITY_COUNT;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = threadId; i < sharedMatrixSize; i += blockDim.x)
    {
        sharedDistanceMatrix[i] = distanceMatrix[i];
    }
    __syncthreads();

    if (idx < POPULATION_SIZE)
    {
        float totalDist = 0.0;
        const int base = idx * CITY_COUNT;
        for (int i = 0; i < CITY_COUNT - 1; i++)
        {
            int c1 = population[base + i];
            int c2 = population[base + i + 1];
            totalDist += sharedDistanceMatrix[c1 * CITY_COUNT + c2];
        }
        // return to start
        int c1 = population[base + CITY_COUNT - 1];
        int c2 = population[base];
        totalDist += sharedDistanceMatrix[c1 * CITY_COUNT + c2];

        fitness[idx] = 1.0 / totalDist;
    }
}

// Kernel: Tournament selection
__global__ void tournamentSelectionKernel(const float *fitness, int *selectedIndices, int POPULATION_SIZE, int tournamentSize, hiprandState_t *states)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprandState_t localState = states[idx];
        float bestFit = -1.0;
        int bestIdx = -1;
        for (int i = 0; i < tournamentSize; i++)
        {
            int cand = hiprand(&localState) % POPULATION_SIZE;
            float f = fitness[cand];
            if (f > bestFit)
            {
                bestFit = f;
                bestIdx = cand;
            }
        }
        selectedIndices[idx] = bestIdx;
        states[idx] = localState;
    }
}

// Kernel: OX Crossover
__global__ void crossoverKernel(const int *population, const int *selectedIndices,
                                int *newPopulation, hiprandState_t *states, int halfNonElite, int CITY_COUNT)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Handle pairs: idx-th pair corresponds to indices 2*idx and 2*idx+1
    if (idx < halfNonElite)
    {
        int parent1Idx = selectedIndices[2 * idx];
        int parent2Idx = selectedIndices[2 * idx + 1];
        const int *p1 = &population[parent1Idx * CITY_COUNT];
        const int *p2 = &population[parent2Idx * CITY_COUNT];

        int *child1 = &newPopulation[(2 * idx) * CITY_COUNT];
        int *child2 = &newPopulation[(2 * idx + 1) * CITY_COUNT];

        hiprandState_t localState = states[idx];
        int start = hiprand(&localState) % CITY_COUNT;
        int end = hiprand(&localState) % CITY_COUNT;
        if (start > end)
        {
            int tmp = start;
            start = end;
            end = tmp;
        }

        // OX crossover:
        // child1
        for (int i = start; i <= end; i++)
            child1[i] = p1[i];
        {
            int cpos = (end + 1) % CITY_COUNT;
            for (int i = 0; i < CITY_COUNT; i++)
            {
                int gene = p2[i];
                bool found = false;
                for (int j = start; j <= end; j++)
                {
                    if (child1[j] == gene)
                    {
                        found = true;
                        break;
                    }
                }
                if (!found)
                {
                    child1[cpos] = gene;
                    cpos = (cpos + 1) % CITY_COUNT;
                }
            }
        }

        // child2
        for (int i = start; i <= end; i++)
            child2[i] = p2[i];
        {
            int cpos = (end + 1) % CITY_COUNT;
            for (int i = 0; i < CITY_COUNT; i++)
            {
                int gene = p1[i];
                bool found = false;
                for (int j = start; j <= end; j++)
                {
                    if (child2[j] == gene)
                    {
                        found = true;
                        break;
                    }
                }
                if (!found)
                {
                    child2[cpos] = gene;
                    cpos = (cpos + 1) % CITY_COUNT;
                }
            }
        }

        states[idx] = localState;
    }
}

// Kernel: Mutation
__global__ void mutationKernel(int *population, float mutationRate, hiprandState_t *states, int POPULATION_SIZE, int CITY_COUNT)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprandState_t localState = states[idx];
        float r = hiprand_uniform(&localState);
        if (r < mutationRate)
        {
            int c1 = hiprand(&localState) % CITY_COUNT;
            int c2 = hiprand(&localState) % CITY_COUNT;
            int base = idx * CITY_COUNT;
            int temp = population[base + c1];
            population[base + c1] = population[base + c2];
            population[base + c2] = temp;
        }
        states[idx] = localState;
    }
}

City *initializeCities()
{
    City *cities = new City[CITY_COUNT];
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dist(0, 100);
    for (int i = 0; i < CITY_COUNT; ++i)
    {
        cities[i] = {dist(gen), dist(gen)};
    }
    return cities;
}

float *computeDistanceMatrix(const City *cities)
{
    float *distMat = new float[CITY_COUNT * CITY_COUNT];
    for (int i = 0; i < CITY_COUNT; ++i)
    {
        for (int j = 0; j < CITY_COUNT; ++j)
        {
            if (i == j)
                distMat[i * CITY_COUNT + j] = 0.0;
            else
            {
                float dx = cities[i].x - cities[j].x;
                float dy = cities[i].y - cities[j].y;
                distMat[i * CITY_COUNT + j] = sqrt(dx * dx + dy * dy);
            }
        }
    }
    return distMat;
}

__global__ void initPopulationKernel(int *d_population, const int *d_template, hiprandState_t *d_states, int POPULATION_SIZE, int CITY_COUNT)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprandState_t localState = d_states[idx];

        int startPos = idx * CITY_COUNT;
        // Copy template
        for (int i = 0; i < CITY_COUNT; i++)
        {
            d_population[startPos + i] = d_template[i];
        }

        // Shuffle (Fisher-Yates)
        for (int i = CITY_COUNT - 1; i > 0; i--)
        {
            int r = hiprand(&localState) % (i + 1);
            int temp = d_population[startPos + i];
            d_population[startPos + i] = d_population[startPos + r];
            d_population[startPos + r] = temp;
        }

        d_states[idx] = localState;
    }
}

void initializePopulationOnGPU(int *d_population, hiprandState_t *d_states)
{
    // Create template array on CPU
    std::vector<int> templatePath(CITY_COUNT);
    std::iota(templatePath.begin(), templatePath.end(), 0);

    // Copy template to GPU
    int *d_template;
    CUDA_CHECK(hipMalloc((void **)&d_template, CITY_COUNT * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_template, templatePath.data(), CITY_COUNT * sizeof(int), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (POPULATION_SIZE + threads - 1) / threads;
    initPopulationKernel<<<blocks, threads>>>(d_population, d_template, d_states, POPULATION_SIZE, CITY_COUNT);
    hipDeviceSynchronize();

    hipFree(d_template);
}

float calculatePathDistance(const int *path, const float *distMat)
{
    float totalDist = 0.0;
    for (int i = 0; i < CITY_COUNT - 1; i++)
    {
        totalDist += distMat[path[i] * CITY_COUNT + path[i + 1]];
    }
    totalDist += distMat[path[CITY_COUNT - 1] * CITY_COUNT + path[0]];
    return totalDist;
}

__host__ void geneticAlgorithm(City* cities, float* h_distMat)
{
    // Allocate distance matrix on GPU
    float *d_distanceMatix;
    CUDA_CHECK(hipMalloc((void **)&d_distanceMatix, CITY_COUNT * CITY_COUNT * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_distanceMatix, h_distMat, CITY_COUNT * CITY_COUNT * sizeof(float), hipMemcpyHostToDevice));

    int *d_population;
    int *d_newPopulation;
    float *d_fitness;
    int *d_selectedIndices;

    CUDA_CHECK(hipMalloc((void **)&d_population, POPULATION_SIZE * CITY_COUNT * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_newPopulation, POPULATION_SIZE * CITY_COUNT * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_fitness, POPULATION_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_selectedIndices, POPULATION_SIZE * sizeof(int)));

    // Setup RNG
    hiprandState_t *d_states;
    CUDA_CHECK(hipMalloc((void **)&d_states, POPULATION_SIZE * sizeof(hiprandState_t)));
    int threads = 256;
    int blocks = (POPULATION_SIZE + threads - 1) / threads;
    initRNGStates<<<blocks, threads>>>(d_states, 1234, POPULATION_SIZE);

    // Initialize population on GPU
    initializePopulationOnGPU(d_population, d_states);

    int eliteCount = (int)(ELITISM_THRESHOLD * POPULATION_SIZE);
    int nonEliteCount = POPULATION_SIZE - eliteCount;
    int halfNonElite = nonEliteCount / 2;
    int cb = (nonEliteCount + threads - 1) / threads;
    int mutation_blocks = (nonEliteCount + threads - 1) / threads;
    int sharedMemSize = CITY_COUNT * CITY_COUNT * sizeof(float);

    float *result_check_fitness = new float[POPULATION_SIZE];

    for (int gen = 0; gen < GENERATIONS; gen++)
    {
        // Compute fitness
        computeFitnessKernel<<<blocks, threads,sharedMemSize>>>(d_population, d_distanceMatix, d_fitness, POPULATION_SIZE, CITY_COUNT);
        hipDeviceSynchronize();

        // Copy fitness to host and do elitism
        float *h_fitness = new float[POPULATION_SIZE];
        CUDA_CHECK(hipMemcpy(h_fitness, d_fitness, POPULATION_SIZE * sizeof(float), hipMemcpyDeviceToHost));

        vector<pair<float, int>> fitIdx(POPULATION_SIZE);
        for (int i = 0; i < POPULATION_SIZE; i++)
            fitIdx[i] = {h_fitness[i], i};
        sort(fitIdx.begin(), fitIdx.end(), greater<>());

        int *h_eliteIndices = new int[eliteCount];
        for (int i = 0; i < eliteCount; i++)
        {
            h_eliteIndices[i] = fitIdx[i].second;
        }

        int *h_popBuffer = new int[POPULATION_SIZE * CITY_COUNT];
        CUDA_CHECK(hipMemcpy(h_popBuffer, d_population, POPULATION_SIZE * CITY_COUNT * sizeof(int), hipMemcpyDeviceToHost));

        int *h_eliteBuffer = new int[eliteCount * CITY_COUNT];
        for (int e = 0; e < eliteCount; e++)
        {
            int idx = h_eliteIndices[e];
            memcpy(&h_eliteBuffer[e * CITY_COUNT], &h_popBuffer[idx * CITY_COUNT], CITY_COUNT * sizeof(int));
        }

        CUDA_CHECK(hipMemcpy(d_newPopulation, h_eliteBuffer, eliteCount * CITY_COUNT * sizeof(int), hipMemcpyHostToDevice));
        hipDeviceSynchronize();

        delete[] h_eliteIndices;
        delete[] h_fitness;
        delete[] h_popBuffer;
        delete[] h_eliteBuffer;

        // Selection
        tournamentSelectionKernel<<<blocks, threads>>>(d_fitness, d_selectedIndices, POPULATION_SIZE, TOURNAMENT_SIZE, d_states);
        hipDeviceSynchronize();

        // Crossover
        crossoverKernel<<<cb, threads>>>(d_population, d_selectedIndices, d_newPopulation + eliteCount * CITY_COUNT, d_states, halfNonElite, CITY_COUNT);
        hipDeviceSynchronize();

        // Mutation
        mutationKernel<<<mutation_blocks, threads>>>(d_newPopulation + (eliteCount * CITY_COUNT), MUTATION_RATE, d_states, nonEliteCount, CITY_COUNT);
        hipDeviceSynchronize();

        int *temp = d_population;
        d_population = d_newPopulation;
        d_newPopulation = temp;

        if (PRINT_EACH_ITERATION && (gen % 50 == 0))
        {
            CUDA_CHECK(hipMemcpy(result_check_fitness, d_fitness, POPULATION_SIZE * sizeof(float), hipMemcpyDeviceToHost));
            float bestFit = result_check_fitness[0];
            for (int i = 1; i < POPULATION_SIZE; i++)
            {
                if (result_check_fitness[i] > bestFit)
                {
                    bestFit = result_check_fitness[i];
                }
            }
            float bestDistance = 1.0 / bestFit;
            cout << "Generation " << gen << ": Best Distance = " << bestDistance << endl;
        }
    }

    // Final calculation
    computeFitnessKernel<<<blocks, threads>>>(d_population, d_distanceMatix, d_fitness, POPULATION_SIZE, CITY_COUNT);
    hipDeviceSynchronize();

    float *h_fitness = new float[POPULATION_SIZE];
    CUDA_CHECK(hipMemcpy(h_fitness, d_fitness, POPULATION_SIZE * sizeof(float), hipMemcpyDeviceToHost));

    int bestIdx = 0;
    float bestFit = h_fitness[0];
    for (int i = 1; i < POPULATION_SIZE; i++)
    {
        if (h_fitness[i] > bestFit)
        {
            bestFit = h_fitness[i];
            bestIdx = i;
        }
    }

    int *h_solution = new int[CITY_COUNT];
    CUDA_CHECK(hipMemcpy(h_solution, d_population + bestIdx * CITY_COUNT, CITY_COUNT * sizeof(int), hipMemcpyDeviceToHost));

    if (PRINT_EACH_ITERATION){
        float bestDistance = 1.0 / bestFit;
        cout << "Best distance: " << bestDistance << endl;
        cout << "Best path:" << endl;
        for (int i = 0; i < CITY_COUNT; i++)
            cout << h_solution[i] << " ";
        cout << "\n";
    }

    delete[] h_solution;
    delete[] h_fitness;
    delete[] result_check_fitness; 

    hipFree(d_population);
    hipFree(d_newPopulation);
    hipFree(d_fitness);
    hipFree(d_selectedIndices);
    hipFree(d_distanceMatix);
    hipFree(d_states);
}


int main(int argc, char* argv[])
{
    if (argc > 1) CITY_COUNT = stoi(argv[1]);
    if (argc > 2) POPULATION_SIZE = stoi(argv[2]);
    if (argc > 3) GENERATIONS = stoi(argv[3]);
    if (argc > 4) {
        string val = argv[4];
        if (val == "false" || val == "0") {
            PRINT_EACH_ITERATION = false;
        } else {
            PRINT_EACH_ITERATION = true;
        }
    }

    if(PRINT_EACH_ITERATION){
        cout << "CITY_COUNT=" << CITY_COUNT << ", POPULATION_SIZE=" << POPULATION_SIZE
            << ", GENERATIONS=" << GENERATIONS
            << ", PRINT_EACH_ITERATION=" << (PRINT_EACH_ITERATION ? "true" : "false") << endl;
    }


    City *cities = initializeCities();
    float *h_distMat = computeDistanceMatrix(cities);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    float ms = 0.0f;

    geneticAlgorithm(cities, h_distMat);

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    
    if(PRINT_EACH_ITERATION){
        cout <<"Time usage:" << ms << " ms" << endl;
    }else{
        cout << ms << endl;
    }

    delete[] h_distMat;
    delete[] cities;

    return 0;
}
