#include <iostream>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <random>
#include <limits>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <string>

#define CUDA_CHECK(call)                                                                          \
    do                                                                                            \
    {                                                                                             \
        hipError_t err = call;                                                                   \
        if (err != hipSuccess)                                                                   \
        {                                                                                         \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
            exit(1);                                                                              \
        }                                                                                         \
    } while (0)

using namespace std;

int CITY_COUNT = 100;                 // city scale
int POPULATION_SIZE = 1000;           // population scale
int GENERATIONS = 1000;               // iteration times
const double MUTATION_RATE = 0.1;     // mutation rate
const double ELITISM_THRESHOLD = 0.2; // elite
bool PRINT_EACH_ITERATION = true;     // debug
const int TOURNAMENT_SIZE = 5;        // For tournament selection
struct City
{
    int x, y;
};

// Kernel to initialize curand states
__global__ void initRNGStates(hiprandState_t *states, unsigned long seed, int POPULATION_SIZE)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

// GPU kernel: compute fitness
__global__ void computeFitnessKernel(const int *population, const double *distanceMatrix, double *fitness, int POPULATION_SIZE, int CITY_COUNT)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        double totalDist = 0.0;
        const int base = idx * CITY_COUNT;
        for (int i = 0; i < CITY_COUNT - 1; i++)
        {
            int c1 = population[base + i];
            int c2 = population[base + i + 1];
            totalDist += distanceMatrix[c1 * CITY_COUNT + c2];
        }
        // return to start
        int c1 = population[base + CITY_COUNT - 1];
        int c2 = population[base];
        totalDist += distanceMatrix[c1 * CITY_COUNT + c2];

        fitness[idx] = 1.0 / totalDist;
    }
}

// Kernel: Tournament selection
__global__ void tournamentSelectionKernel(const double *fitness, int *selectedIndices, int POPULATION_SIZE, int tournamentSize, hiprandState_t *states)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprandState_t localState = states[idx];
        double bestFit = -1.0;
        int bestIdx = -1;
        for (int i = 0; i < tournamentSize; i++)
        {
            int cand = hiprand(&localState) % POPULATION_SIZE;
            double f = fitness[cand];
            if (f > bestFit)
            {
                bestFit = f;
                bestIdx = cand;
            }
        }
        selectedIndices[idx] = bestIdx;
        states[idx] = localState;
    }
}

// Kernel: OX Crossover
__global__ void crossoverKernel(const int *population, const int *selectedIndices,
                                int *newPopulation, hiprandState_t *states, int halfNonElite, int CITY_COUNT)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Handle pairs: idx-th pair corresponds to indices 2*idx and 2*idx+1
    if (idx < halfNonElite)
    {
        int parent1Idx = selectedIndices[2 * idx];
        int parent2Idx = selectedIndices[2 * idx + 1];
        const int *p1 = &population[parent1Idx * CITY_COUNT];
        const int *p2 = &population[parent2Idx * CITY_COUNT];

        int *child1 = &newPopulation[(2 * idx) * CITY_COUNT];
        int *child2 = &newPopulation[(2 * idx + 1) * CITY_COUNT];

        hiprandState_t localState = states[idx];
        int start = hiprand(&localState) % CITY_COUNT;
        int end = hiprand(&localState) % CITY_COUNT;
        if (start > end)
        {
            int tmp = start;
            start = end;
            end = tmp;
        }

        // OX crossover:
        // child1
        for (int i = start; i <= end; i++)
            child1[i] = p1[i];
        {
            int cpos = (end + 1) % CITY_COUNT;
            for (int i = 0; i < CITY_COUNT; i++)
            {
                int gene = p2[i];
                bool found = false;
                for (int j = start; j <= end; j++)
                {
                    if (child1[j] == gene)
                    {
                        found = true;
                        break;
                    }
                }
                if (!found)
                {
                    child1[cpos] = gene;
                    cpos = (cpos + 1) % CITY_COUNT;
                }
            }
        }

        // child2
        for (int i = start; i <= end; i++)
            child2[i] = p2[i];
        {
            int cpos = (end + 1) % CITY_COUNT;
            for (int i = 0; i < CITY_COUNT; i++)
            {
                int gene = p1[i];
                bool found = false;
                for (int j = start; j <= end; j++)
                {
                    if (child2[j] == gene)
                    {
                        found = true;
                        break;
                    }
                }
                if (!found)
                {
                    child2[cpos] = gene;
                    cpos = (cpos + 1) % CITY_COUNT;
                }
            }
        }

        states[idx] = localState;
    }
}

// Kernel: Mutation
__global__ void mutationKernel(int *population, double mutationRate, hiprandState_t *states, int POPULATION_SIZE, int CITY_COUNT)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprandState_t localState = states[idx];
        float r = hiprand_uniform(&localState);
        if (r < mutationRate)
        {
            int c1 = hiprand(&localState) % CITY_COUNT;
            int c2 = hiprand(&localState) % CITY_COUNT;
            int base = idx * CITY_COUNT;
            int temp = population[base + c1];
            population[base + c1] = population[base + c2];
            population[base + c2] = temp;
        }
        states[idx] = localState;
    }
}

City *initializeCities()
{
    City *cities = new City[CITY_COUNT];
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dist(0, 100);
    for (int i = 0; i < CITY_COUNT; ++i)
    {
        cities[i] = {dist(gen), dist(gen)};
    }
    return cities;
}

double *computeDistanceMatrix(const City *cities)
{
    double *distMat = new double[CITY_COUNT * CITY_COUNT];
    for (int i = 0; i < CITY_COUNT; ++i)
    {
        for (int j = 0; j < CITY_COUNT; ++j)
        {
            if (i == j)
                distMat[i * CITY_COUNT + j] = 0.0;
            else
            {
                double dx = cities[i].x - cities[j].x;
                double dy = cities[i].y - cities[j].y;
                distMat[i * CITY_COUNT + j] = sqrt(dx * dx + dy * dy);
            }
        }
    }
    return distMat;
}

__global__ void initPopulationKernel(int *d_population, const int *d_template, hiprandState_t *d_states, int POPULATION_SIZE, int CITY_COUNT)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < POPULATION_SIZE)
    {
        hiprandState_t localState = d_states[idx];

        int startPos = idx * CITY_COUNT;
        // Copy template
        for (int i = 0; i < CITY_COUNT; i++)
        {
            d_population[startPos + i] = d_template[i];
        }

        // Shuffle (Fisher-Yates)
        for (int i = CITY_COUNT - 1; i > 0; i--)
        {
            int r = hiprand(&localState) % (i + 1);
            int temp = d_population[startPos + i];
            d_population[startPos + i] = d_population[startPos + r];
            d_population[startPos + r] = temp;
        }

        d_states[idx] = localState;
    }
}

void initializePopulationOnGPU(int *d_population, hiprandState_t *d_states)
{
    // Create template array on CPU
    std::vector<int> templatePath(CITY_COUNT);
    std::iota(templatePath.begin(), templatePath.end(), 0);

    // Copy template to GPU
    int *d_template;
    CUDA_CHECK(hipMalloc((void **)&d_template, CITY_COUNT * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_template, templatePath.data(), CITY_COUNT * sizeof(int), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (POPULATION_SIZE + threads - 1) / threads;
    initPopulationKernel<<<blocks, threads>>>(d_population, d_template, d_states, POPULATION_SIZE, CITY_COUNT);
    hipDeviceSynchronize();

    hipFree(d_template);
}

double calculatePathDistance(const int *path, const double *distMat)
{
    double totalDist = 0.0;
    for (int i = 0; i < CITY_COUNT - 1; i++)
    {
        totalDist += distMat[path[i] * CITY_COUNT + path[i + 1]];
    }
    totalDist += distMat[path[CITY_COUNT - 1] * CITY_COUNT + path[0]];
    return totalDist;
}

__host__ void geneticAlgorithm(City* cities, double* h_distMat)
{
    // Allocate distance matrix on GPU
    double *d_distanceMatix;
    CUDA_CHECK(hipMalloc((void **)&d_distanceMatix, CITY_COUNT * CITY_COUNT * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_distanceMatix, h_distMat, CITY_COUNT * CITY_COUNT * sizeof(double), hipMemcpyHostToDevice));

    int *d_population;
    int *d_newPopulation;
    double *d_fitness;
    int *d_selectedIndices;

    CUDA_CHECK(hipMalloc((void **)&d_population, POPULATION_SIZE * CITY_COUNT * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_newPopulation, POPULATION_SIZE * CITY_COUNT * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_fitness, POPULATION_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc((void **)&d_selectedIndices, POPULATION_SIZE * sizeof(int)));

    // Setup RNG
    hiprandState_t *d_states;
    CUDA_CHECK(hipMalloc((void **)&d_states, POPULATION_SIZE * sizeof(hiprandState_t)));
    int threads = 256;
    int blocks = (POPULATION_SIZE + threads - 1) / threads;
    initRNGStates<<<blocks, threads>>>(d_states, 1234, POPULATION_SIZE);

    // Initialize population on GPU
    initializePopulationOnGPU(d_population, d_states);

    int eliteCount = (int)(ELITISM_THRESHOLD * POPULATION_SIZE);
    int nonEliteCount = POPULATION_SIZE - eliteCount;
    int halfNonElite = nonEliteCount / 2;
    int cb = (nonEliteCount + threads - 1) / threads;
    int mutation_blocks = (nonEliteCount + threads - 1) / threads;


    double *result_check_fitness = new double[POPULATION_SIZE];

    cout << "finish initialization" << endl;
    for (int gen = 0; gen < GENERATIONS; gen++)
    {
        // Compute fitness
        computeFitnessKernel<<<blocks, threads>>>(d_population, d_distanceMatix, d_fitness, POPULATION_SIZE, CITY_COUNT);
        hipDeviceSynchronize();

        // Copy fitness to host and do elitism
        double *h_fitness = new double[POPULATION_SIZE];
        CUDA_CHECK(hipMemcpy(h_fitness, d_fitness, POPULATION_SIZE * sizeof(double), hipMemcpyDeviceToHost));

        vector<pair<double, int>> fitIdx(POPULATION_SIZE);
        for (int i = 0; i < POPULATION_SIZE; i++)
            fitIdx[i] = {h_fitness[i], i};
        sort(fitIdx.begin(), fitIdx.end(), greater<>());

        int *h_eliteIndices = new int[eliteCount];
        for (int i = 0; i < eliteCount; i++)
        {
            h_eliteIndices[i] = fitIdx[i].second;
        }

        int *h_popBuffer = new int[POPULATION_SIZE * CITY_COUNT];
        CUDA_CHECK(hipMemcpy(h_popBuffer, d_population, POPULATION_SIZE * CITY_COUNT * sizeof(int), hipMemcpyDeviceToHost));

        int *h_eliteBuffer = new int[eliteCount * CITY_COUNT];
        for (int e = 0; e < eliteCount; e++)
        {
            int idx = h_eliteIndices[e];
            memcpy(&h_eliteBuffer[e * CITY_COUNT], &h_popBuffer[idx * CITY_COUNT], CITY_COUNT * sizeof(int));
        }

        CUDA_CHECK(hipMemcpy(d_newPopulation, h_eliteBuffer, eliteCount * CITY_COUNT * sizeof(int), hipMemcpyHostToDevice));
        hipDeviceSynchronize();

        delete[] h_eliteIndices;
        delete[] h_fitness;
        delete[] h_popBuffer;
        delete[] h_eliteBuffer;

        // Selection
        tournamentSelectionKernel<<<blocks, threads>>>(d_fitness, d_selectedIndices, POPULATION_SIZE, TOURNAMENT_SIZE, d_states);
        hipDeviceSynchronize();

        // Crossover
        crossoverKernel<<<cb, threads>>>(d_population, d_selectedIndices, d_newPopulation + eliteCount * CITY_COUNT, d_states, halfNonElite, CITY_COUNT);
        hipDeviceSynchronize();

        // Mutation
        mutationKernel<<<mutation_blocks, threads>>>(d_newPopulation + (eliteCount * CITY_COUNT), MUTATION_RATE, d_states, nonEliteCount, CITY_COUNT);
        hipDeviceSynchronize();

        int *temp = d_population;
        d_population = d_newPopulation;
        d_newPopulation = temp;

        if (PRINT_EACH_ITERATION && (gen % 50 == 0))
        {
            CUDA_CHECK(hipMemcpy(result_check_fitness, d_fitness, POPULATION_SIZE * sizeof(double), hipMemcpyDeviceToHost));
            double bestFit = result_check_fitness[0];
            for (int i = 1; i < POPULATION_SIZE; i++)
            {
                if (result_check_fitness[i] > bestFit)
                {
                    bestFit = result_check_fitness[i];
                }
            }
            double bestDistance = 1.0 / bestFit;
            cout << "Generation " << gen << ": Best Distance = " << bestDistance << endl;
        }
    }

    // Final calculation
    computeFitnessKernel<<<blocks, threads>>>(d_population, d_distanceMatix, d_fitness, POPULATION_SIZE, CITY_COUNT);
    hipDeviceSynchronize();

    double *h_fitness = new double[POPULATION_SIZE];
    CUDA_CHECK(hipMemcpy(h_fitness, d_fitness, POPULATION_SIZE * sizeof(double), hipMemcpyDeviceToHost));

    int bestIdx = 0;
    double bestFit = h_fitness[0];
    for (int i = 1; i < POPULATION_SIZE; i++)
    {
        if (h_fitness[i] > bestFit)
        {
            bestFit = h_fitness[i];
            bestIdx = i;
        }
    }

    int *h_solution = new int[CITY_COUNT];
    CUDA_CHECK(hipMemcpy(h_solution, d_population + bestIdx * CITY_COUNT, CITY_COUNT * sizeof(int), hipMemcpyDeviceToHost));

    double bestDistance = 1.0 / bestFit;
    cout << "Best distance: " << bestDistance << endl;
    cout << "Best path:" << endl;
    for (int i = 0; i < CITY_COUNT; i++)
        cout << h_solution[i] << " ";
    cout << "\n";

    // Cleanup
    delete[] h_solution;
    delete[] h_fitness;
    delete[] result_check_fitness; 
    // Removed delete[] cities; from here to avoid double free.

    hipFree(d_population);
    hipFree(d_newPopulation);
    hipFree(d_fitness);
    hipFree(d_selectedIndices);
    hipFree(d_distanceMatix);
    hipFree(d_states);
}


int main(int argc, char* argv[])
{
    if (argc > 1) CITY_COUNT = stoi(argv[1]);
    if (argc > 2) POPULATION_SIZE = stoi(argv[2]);
    if (argc > 3) GENERATIONS = stoi(argv[3]);
    if (argc > 4) {
        string val = argv[4];
        if (val == "false" || val == "0") {
            PRINT_EACH_ITERATION = false;
        } else {
            PRINT_EACH_ITERATION = true;
        }
    }

    cout << "CITY_COUNT=" << CITY_COUNT << ", POPULATION_SIZE=" << POPULATION_SIZE
         << ", GENERATIONS=" << GENERATIONS
         << ", PRINT_EACH_ITERATION=" << (PRINT_EACH_ITERATION ? "true" : "false") << endl;

    City *cities = initializeCities();
    double *h_distMat = computeDistanceMatrix(cities);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    float ms = 0.0f;

    geneticAlgorithm(cities, h_distMat);

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    cout << "Time (ms): " << ms << endl;

    delete[] h_distMat;
    delete[] cities;

    return 0;
}
